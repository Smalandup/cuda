#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <common/utils.h>

#include "daubechies4.h"

void cpu_d4_transform(double *src, double* dest, const int n)
{
    
    if (n >= 4) 
    {
        int i=0,j=0;
        const int half = n>>1;

        for (i = 0; i < half; i++) 
        {
            j = 2*i;
            if (j < n-3) {
                dest[i]      = src[j]*_h[0] + src[j+1]*_h[1] + src[j+2]*_h[2] + src[j+3]*_h[3];
                dest[i+half] = src[j]*_g[0] + src[j+1]*_g[1] + src[j+2]*_g[2] + src[j+3]*_g[3];
            } 
            else { 
                break; 
            }
        }

        dest[i]      = src[n-2]*_h[0] + src[n-1]*_h[1] + src[0]*_h[2] + src[1]*_h[3];
        dest[i+half] = src[n-2]*_g[0] + src[n-1]*_g[1] + src[0]*_g[2] + src[1]*_g[3];
    }
}

void cpu_d4_inv_transform(double *src, double *dest, int n)
{

    if(n >= 4)
    {
        int i=0,j;
        int half = n >> 1;
       
        dest[0] = src[half-1]*_ih[0] + src[n-1]*_ih[1] + src[0]*_ih[2] + src[half]*_ih[3];
        dest[1] = src[half-1]*_ig[0] + src[n-1]*_ig[1] + src[0]*_ig[2] + src[half]*_ig[3];
        j = 2;
        for (;i < half-1; i++) 
        { 
          dest[j++]    = src[i]*_ih[0] + src[i+half]*_ih[1] + src[i+1]*_ih[2] + src[i+half+1]*_ih[3];
          dest[j++]    = src[i]*_ig[0] + src[i+half]*_ig[1] + src[i+1]*_ig[2] + src[i+half+1]*_ig[3];
        }
    }
}

/*  The Daubechies-4 wavelet forward pass
    I adapted this code from http://bearcave.com/misl/misl_tech/wavelets/index.html
    To compute the full the full wavelet transform of a signal of size N
    We call this kernel log_2(N) times (assuming N is power of 2) */

__global__ void gpu_dwt_pass(double *src, double *dest, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int half = n >> 1;

    if(2*i < (n-3)) {
        dest[i]             = src[2*i]*h[0] + src[2*i+1]*h[1] + src[2*i+2]*h[2] + src[2*i+3]*h[3];
        dest[i+half]        = src[2*i]*g[0] + src[2*i+1]*g[1] + src[2*i+2]*g[2] + src[2*i+3]*g[3];
    }
    if(2*i == (n-2)) {
        dest[i]         = src[n-2]*h[0] + src[n-1]*h[1] + src[0]*h[2] + src[1]*h[3];
        dest[i+half]    = src[n-2]*g[0] + src[n-1]*g[1] + src[0]*g[2] + src[1]*g[3];
    }

}

__global__ void gpu_idwt_pass(double *src, double *dest, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int half = n >> 1;

    if(i == 0)
    {
        dest[0] = src[half-1]*ih[0] + src[n-1]*ih[1] + src[0]*ih[2] + src[half]*ih[3];
        dest[1] = src[half-1]*ig[0] + src[n-1]*ig[1] + src[0]*ig[2] + src[half]*ig[3];
    } 
    if (i < (half-1)) 
    {
        dest[2*i+2]    = src[i]*ih[0] + src[i+half]*ih[1] + src[i+1]*ih[2] + src[i+half+1]*ih[3];
        dest[2*i+3]    = src[i]*ig[0] + src[i+half]*ig[1] + src[i+1]*ig[2] + src[i+half+1]*ig[3];
    }
}

double gpu_dwt(double *t, int N)
{
    assert(check_power_two(N));

    size_t size = N*sizeof(double);
    double *d_src,*d_dst;
    int n = N;

    int threadsPerBlock = 512;
    int blocksPerGrid =(N + threadsPerBlock - 1) / threadsPerBlock;

    CUDA_CALL(hipMalloc((void**)&d_src,size));
    CUDA_CALL(hipMalloc((void**)&d_dst,size));

    CUDA_CALL(hipMemcpy(d_src,t,size,hipMemcpyHostToDevice));

    clock_t begin, end;

    begin = clock();
    while(n >= 4)
    {
        gpu_dwt_pass<<<blocksPerGrid,threadsPerBlock>>>(d_src,d_dst,n);
        // we need only copy the n first elements, not the whole signal
        CUDA_CALL(hipMemcpy(d_src,d_dst,n*sizeof(double),hipMemcpyDeviceToDevice)); 
        n = n>>1;
    }
    hipDeviceSynchronize();
    end = clock();
    CUDA_CALL(hipMemcpy(t,d_src,size,hipMemcpyDeviceToHost));
    
    printf("GPU Elapsed: %lfs \n", elapsed(begin,end));

    CUDA_CALL(hipFree(d_src));
    CUDA_CALL(hipFree(d_dst));
    return elapsed(begin,end);
}

double gpu_idwt(double *t, int N)
{
    assert(check_power_two(N));

    size_t size = N*sizeof(double);
    double *d_src,*d_dst;
    int n = 4;

    int threadsPerBlock = 512;
    int blocksPerGrid =(N + threadsPerBlock - 1) / threadsPerBlock;

    CUDA_CALL(hipMalloc((void**)&d_src,size));
    CUDA_CALL(hipMalloc((void**)&d_dst,size));

    CUDA_CALL(hipMemcpy(d_src,t,size,hipMemcpyHostToDevice));

    clock_t begin, end;

    begin = clock();
    while(n <= N)
    {
        gpu_idwt_pass<<<blocksPerGrid,threadsPerBlock>>>(d_src,d_dst,n);
        CUDA_CALL(hipMemcpy(d_src,d_dst,n*sizeof(double),hipMemcpyDeviceToDevice));
        n = n << 1;
    }
    hipDeviceSynchronize();
    end = clock();
    CUDA_CALL(hipMemcpy(t,d_src,size,hipMemcpyDeviceToHost));
    
    printf("GPU Elapsed: %lfs \n", elapsed(begin,end));

    CUDA_CALL(hipFree(d_src));
    CUDA_CALL(hipFree(d_dst));
    return 0;
}

double cpu_dwt(double* t, int N)
{
    assert(check_power_two(N));
    int n=N;
    clock_t begin,end;
    double *tmp = (double*)malloc(N*sizeof(double));

    if(!tmp)
    {
        fprintf(stderr,"(host) cannot allocate memory for DWT\n");
        exit(EXIT_FAILURE);
    }

    begin = clock();
    while(n >= 4) 
    {
        cpu_d4_transform(t,tmp,n);
        memcpy(t,tmp,n*sizeof(double));

        n >>= 1;
    }

    end = clock();
    printf("CPU Elapsed: %lfs\n", elapsed(begin,end));
    free(tmp);
    return elapsed(begin,end);
}

double cpu_idwt(double *t, int N)
{
    assert(check_power_two(N));
    int n;
    clock_t begin, end; 

    double *tmp = (double*)malloc(N*sizeof(double));

    if(!tmp)
    {
        fprintf(stderr,"(host) cannot allocate memory for DWT\n");
        exit(EXIT_FAILURE);
    }

    begin = clock();
    for(n = 4; n <= N; n <<= 1)
    {
        cpu_d4_inv_transform(t,tmp,n);
        memcpy(t,tmp,n*sizeof(double));
    }
    end = clock();

    printf("CPU Elapsed: %lfs\n", elapsed(begin,end));
    free(tmp);

    return elapsed(begin,end);
}

int save_timing_forward()
{
    /*  benchmark CPU v. GPU on forward discrete wavelet transform 
        and save into csv file */

    FILE *save = fopen("results/timing.csv", "w+");

    if(!save) 
    {
        fprintf(stderr, "%s\n", "(host) unable to create timing file..");
        exit(EXIT_FAILURE);
    }

    fprintf(save,"%s,%s,%s\n", "N", "CPU_Time","GPU_Time");

    int n = 1<<10;
    double cpu_time,gpu_time;

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(g),_g,4*sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(h),_h,4*sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ig),_ig,4*sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ih),_ih,4*sizeof(double)));

    while(n <= (1<<24)) 
    {
        printf("n=%d\n",n);

        size_t size = n*sizeof(double);

        double * gpu_coef   = (double*)malloc(size);
        double * cpu_coef   = (double*)malloc(size);
        double * x0         = (double*)malloc(size);

        if(!gpu_coef || !cpu_coef || !x0) {
            fprintf(stderr, "%s\n", "could not allocate memory for signals!\n");
            exit(EXIT_FAILURE);
        }

        /* copy constants */

        fill_rand(x0, n);
        memcpy(gpu_coef,x0,size);
        memcpy(cpu_coef,x0,size); // save initial (random) array 

        gpu_time = gpu_dwt(gpu_coef,n);
        cpu_time = cpu_dwt(cpu_coef,n);

        if(!test_arrays_equal(gpu_coef,cpu_coef,n))
        {
            printf("Arrays not equal!\n");
            exit(EXIT_FAILURE);
        }

        fprintf(save, "%d,%lf,%lf\n", n,cpu_time,gpu_time);

        n <<= 1;

        free(cpu_coef);
        free(gpu_coef);
        free(x0);
    }

    fclose(save);
    return 0;
}

int test_dwt(const int N)
{
    double *signal = (double*)malloc(N*sizeof(double));

    fill_rand(signal, N);

    double * cpu_coef = (double*)malloc(N*sizeof(double));
    double * gpu_coef = (double*)malloc(N*sizeof(double));

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(g),_g,4*sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(h),_h,4*sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ig),_ig,4*sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ih),_ih,4*sizeof(double)));

    memcpy(cpu_coef,signal,N*sizeof(double));
    memcpy(gpu_coef,signal,N*sizeof(double));

    cpu_dwt(cpu_coef,N);
    gpu_dwt(gpu_coef,N);

    if(!test_arrays_equal(cpu_coef,gpu_coef,N))
    {
        printf("DWT not the same on CPU and GPU.\n");
        exit(EXIT_FAILURE);
    }
    printf("DWT test: pass.\n");
    return 0;
}

int test_idwt_cpu(const int N)
{
    double *signal = (double*)malloc(N*sizeof(double));

    fill_rand(signal, N);

    double * cpu_coef = (double*)malloc(N*sizeof(double));
    memcpy(cpu_coef,signal,N*sizeof(double));

    cpu_dwt(cpu_coef,N);
    cpu_idwt(cpu_coef,N);

    if(!test_arrays_equal(cpu_coef,signal,N))
    {
        printf("IDWT fail: signal not reconstructed on CPU.\n");
        exit(EXIT_FAILURE);
    }
    printf("IDWT CPU: pass.\n");
    return 0;
}

int test_idwt_gpu(const int N)
{
    double *signal = (double*)malloc(N*sizeof(double));

    fill_rand(signal, N);

    double * gpu_coef = (double*)malloc(N*sizeof(double));

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(g),_g,4*sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(h),_h,4*sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ig),_ig,4*sizeof(double)));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ih),_ih,4*sizeof(double)));

    memcpy(gpu_coef,signal,N*sizeof(double));

    gpu_dwt(gpu_coef,N);
    gpu_idwt(gpu_coef,N);

    if(!test_arrays_equal(gpu_coef,signal,N))
    {
        printf("IDWT fail: signal not reconstructed on GPU.\n");
        exit(EXIT_FAILURE);
    }
    printf("IDWT GPU: pass.\n");
    return 0;
}

int main()
{
    int N = 1<<19;

    test_dwt(N);
    test_idwt_cpu(N);
    test_idwt_gpu(N);

    save_timing_forward();

    return 0;
}