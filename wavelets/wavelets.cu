#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <common/utils.h>

#include "daubechies4.h"


/*  The Daubechies-4 wavelet forward pass
    I adapted this code from http://bearcave.com/misl/misl_tech/wavelets/index.html
    To compute the full the full wavelet transform of a signal of size N
    We call this kernel log_2(N) times (assuming N is power of 2) */

__global__ void gpu_dwt_pass(double *src, double *dest, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int half = n >> 1;

    if(2*i < (n-3)) {
        dest[i]             = src[2*i]*h[0] + src[2*i+1]*h[1] + src[2*i+2]*h[2] + src[2*i+3]*h[3];
        dest[i+half]        = src[2*i]*g[0] + src[2*i+1]*g[1] + src[2*i+2]*g[2] + src[2*i+3]*g[3];
    }
    if(2*i == (n-2)) {
        dest[i]         = src[n-2]*h[0] + src[n-1]*h[1] + src[0]*h[2] + src[1]*h[3];
        dest[i+half]    = src[n-2]*g[0] + src[n-1]*g[1] + src[0]*g[2] + src[1]*g[3];
    }

}


int gpu_dwt(double *t, int n)
{
    assert(check_power_two(n));

    size_t size = n*sizeof(double);
    double *d_src,*d_dst;

    int threadsPerBlock = 512;
    int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;

    CUDA_CALL(hipMalloc((void**)&d_src,size));
    CUDA_CALL(hipMalloc((void**)&d_dst,size));

    CUDA_CALL(hipMemcpy(d_src,t,size,hipMemcpyHostToDevice));

    clock_t begin, end;

    begin = clock();
    while(n >= 4)
    {
        gpu_dwt_pass<<<blocksPerGrid,threadsPerBlock>>>(d_src,d_dst,n);
        CUDA_CALL(hipMemcpy(d_src,d_dst,size,hipMemcpyDeviceToDevice));
        n = n>>1;
    }
    end = clock();
    CUDA_CALL(hipMemcpy(t,d_src,size,hipMemcpyDeviceToHost));
    
    printf("GPU Elapsed: %lfs \n", elapsed(begin,end));
    return 0;
}

void cpu_d4_transform(double *t, const int n)
{
    
    if (n >= 4) 
    {
        int i=0,j=0;
        const int half = n>>1;

        double * tmp = (double*)malloc(sizeof(double)*n);

        if(!tmp) 
        {
            fprintf(stderr, "cannot allocate memory for daubechies transform");
            exit(EXIT_FAILURE);
        }

        for (i = 0; i < half; i++) 
        {
            j = 2*i;
            if (j < n-3) {
                tmp[i]      = t[j]*_h[0] + t[j+1]*_h[1] + t[j+2]*_h[2] + t[j+3]*_h[3];
                tmp[i+half] = t[j]*_g[0] + t[j+1]*_g[1] + t[j+2]*_g[2] + t[j+3]*_g[3];
            } 
            else { 
                break; 
            }
        }

        tmp[i]      = t[n-2]*_h[0] + t[n-1]*_h[1] + t[0]*_h[2] + t[1]*_h[3];
        tmp[i+half] = t[n-2]*_g[0] + t[n-1]*_g[1] + t[0]*_g[2] + t[1]*_g[3];

        memcpy(t,tmp,n*sizeof(double));
        free(tmp);
    }
}

void cpu_d4_inv_transform(double *t, int n)
{
    if(n >= 4)
    {
        int i=0,j;
        int half = n >> 1;

        double * tmp = (double*)malloc(sizeof(double)*n);

        if(!tmp) 
        {
            fprintf(stderr, "cannot allocate memory for daubechies transform");
            exit(EXIT_FAILURE);
        }

       
        tmp[0] = t[half-1]*_ih[0] + t[n-1]*_ih[1] + t[0]*_ih[2] + t[half]*_ih[3];
        tmp[1] = t[half-1]*_ig[0] + t[n-1]*_ig[1] + t[0]*_ig[2] + t[half]*_ig[3];
        j = 2;
        for (;i < half-1; i++) 
        { 
          tmp[j++]    = t[i]*_ih[0] + t[i+half]*_ih[1] + t[i+1]*_ih[2] + t[i+half+1]*_ih[3];
          tmp[j++]    = t[i]*_ig[0] + t[i+half]*_ig[1] + t[i+1]*_ig[2] + t[i+half+1]*_ig[3];
        }

        memcpy(t,tmp,n*sizeof(double));
        free(tmp);
    }
}



void cpu_dwt(double* t, int N)
{
    assert(check_power_two(N));
    int n=N;

    while(n >= 4) 
    {
        cpu_d4_transform(t,n);
        n >>= 1;
    }
}

void cpu_idwt(double *t, int N)
{
    assert(check_power_two(N));

    int n;

    for(n = 4; n <= N; n <<= 1)
    {
        cpu_d4_inv_transform(t,n);
    }
}

int main()
{
    const int N = (1<<16);
    size_t size = N*sizeof(double);
    clock_t begin, end; 
    double * gpu_coef   = (double*)malloc(size);
    double * cpu_coef   = (double*)malloc(size);
    double * x0         = (double*)malloc(size);

    if(!gpu_coef || !cpu_coef || !x0) {
        fprintf(stderr, "%s\n", "could not allocate memory for signals!\n");
        exit(EXIT_FAILURE);
    }

    fill_rand(x0, N);
    memcpy(gpu_coef,x0,size);
    memcpy(cpu_coef,x0,size); // save initial (random) array 

    gpu_dwt(gpu_coef,N);

    begin = clock();
    cpu_dwt(cpu_coef,N);
    end = clock();

    printf("CPU elapsed: %lfs\n", elapsed(begin,end));

    /* Test wavelet decomposition */

    if(test_arrays_equal(gpu_coef,cpu_coef,N)) 
    {
        printf("Wavelet decomposition is the same on CPU and GPU.\n");
    } else {
        printf("Wavelet decompsition is not the same on CPU and GPU!\n");
        exit(EXIT_FAILURE);
    }

    cpu_idwt(cpu_coef,N);

    if(test_arrays_equal(cpu_coef,x0,N)) 
    {
        printf("Inverse wavelet transform on CPU: success!\n");
    }
}